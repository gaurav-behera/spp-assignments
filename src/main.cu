
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <memory>
#include <cstdint>
#include <filesystem>
#include <string>
#include <unistd.h>
#include <fcntl.h>
#include <sys/mman.h>

namespace solution
{
#define CUDA_ERROR_CHECK(ans)                          \
        {                                              \
                cudaAssert((ans), __FILE__, __LINE__); \
        }
        inline void cudaAssert(hipError_t code, const char *file, int line, bool abort = true)
        {
                if (code != hipSuccess)
                {
                        fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
                        if (abort)
                                exit(code);
                }
        }

        #define TILE_WIDTH 16
        __global__ void convolution2D(float *img_d, float *kernel_d, float* result_d, int n, int gpu_id, int gpu_count)
        {
                __shared__ float img_s[TILE_WIDTH][TILE_WIDTH];
                __shared__ float kernel_s[3][3];

                int tx = threadIdx.x, ty = threadIdx.y;
                int col = blockIdx.x * blockDim.x + tx;
                int row = blockIdx.y * blockDim.y + ty + gpu_id*n/gpu_count;
                if (row < n && col < n)
                {
                        if (tx < 3 && ty < 3)
                                kernel_s[tx][ty] = kernel_d[tx*3+ty];
                        img_s[tx][ty] = img_d[row*n+col];
                        __syncthreads();
                        float sum = 0.0;
                        for(int di = -1; di <= 1; di++)
                        {
                                for(int dj = -1; dj <= 1; dj++) 
                                {
                                        int ni = ty + di, nj = tx + dj;
                                        if (row+di >= 0 && col+dj >= 0 && row+di < n && col+dj < n)
                                        {
                                                if(ni >= 0 and ni < TILE_WIDTH and nj >= 0 and nj < TILE_WIDTH) 
                                                {
                                                        sum += kernel_s[di+1][dj+1] * img_s[ni][nj];
                                                }
                                                else
                                                {
                                                        sum += kernel_s[di+1][dj+1] * img_d[(row+di) * n + (col+dj)];
                                                }
                                                
                                        }
                                }
                        }
                        result_d[(row - gpu_id*n/gpu_count)*n+col] = sum;
                        // result_d[row*n+col] = img_d[row*n+col];
                }
        }

        std::string compute(const std::string &bitmap_path, const float kernel[3][3], const std::int32_t num_rows, const std::int32_t num_cols)
        {
                std::string sol_path = std::filesystem::temp_directory_path() / "student_sol.bmp";

                int size = num_rows * num_cols;

                int bitmap_fd = open(bitmap_path.c_str(), O_RDONLY);
                float *img = static_cast<float *>(mmap(NULL, sizeof(float) * size, PROT_READ, MAP_PRIVATE, bitmap_fd, 0));

                int result_fd = open(sol_path.c_str(), O_CREAT | O_RDWR, 0644);
                ftruncate(result_fd, sizeof(float) * size);
                float *result = reinterpret_cast<float *>(mmap(NULL, sizeof(float) * size, PROT_WRITE | PROT_READ, MAP_SHARED, result_fd, 0));

                float kernel_flat[9];
                for (int i = 0; i < 9; i++)
                {
                        kernel_flat[i] = kernel[i/3][i%3];
                }
                int gpu_count = 4;
                for (int gpu_id = 0; gpu_id < gpu_count; gpu_id++)
                {
                        hipSetDevice(gpu_id);
                        float *img_d, *kernel_d, *result_d;
                        CUDA_ERROR_CHECK(hipMalloc((void**)&img_d, size * sizeof(float)));
                        CUDA_ERROR_CHECK(hipMemcpy(img_d, img, size * sizeof(float), hipMemcpyHostToDevice));
        
                        CUDA_ERROR_CHECK(hipMalloc((void**)&kernel_d, 9 * sizeof(float)));
                        CUDA_ERROR_CHECK(hipMemcpy(kernel_d, kernel_flat, 9 * sizeof(float), hipMemcpyHostToDevice));
        
                        CUDA_ERROR_CHECK(hipMalloc((void **)&result_d, size * sizeof(float)/gpu_count));
        
                        dim3 DimGrid(num_rows / (gpu_count * TILE_WIDTH), num_cols / TILE_WIDTH, 1);
                        dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);

                        convolution2D<<<DimGrid, DimBlock>>>(img_d, kernel_d, result_d, num_cols, gpu_id, gpu_count);
                        
                        hipDeviceSynchronize();
                        
                        CUDA_ERROR_CHECK(hipMemcpy(result + gpu_id*size/gpu_count, result_d, size * sizeof(float) / gpu_count, hipMemcpyDeviceToHost));
                }


                return sol_path;
        }
};