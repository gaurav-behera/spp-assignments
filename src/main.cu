
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <memory>
#include <cstdint>
#include <filesystem>
#include <string>
#include <unistd.h>
#include <fcntl.h>
#include <sys/mman.h>

namespace solution
{
#define CUDA_ERROR_CHECK(ans)                          \
        {                                              \
                cudaAssert((ans), __FILE__, __LINE__); \
        }
        inline void cudaAssert(hipError_t code, const char *file, int line, bool abort = true)
        {
                if (code != hipSuccess)
                {
                        fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
                        if (abort)
                                exit(code);
                }
        }

        __global__ void convolution2D(float *img_d, float *kernel_d, float* result_d, int n)
        {
                int i = blockIdx.x * blockDim.x + threadIdx.x;
                // printf("Hello from the GPU\n");
                if (i < n)
                {
                        result_d[i] = img_d[i];
                }
        }

        std::string compute(const std::string &bitmap_path, const float kernel[3][3], const std::int32_t num_rows, const std::int32_t num_cols)
        {
                std::string sol_path = std::filesystem::temp_directory_path() / "student_sol.bmp";

                int size = num_rows * num_cols;

                int bitmap_fd = open(bitmap_path.c_str(), O_RDONLY);
		float *img = static_cast<float *>(mmap(NULL, sizeof(float) * size, PROT_READ, MAP_PRIVATE, bitmap_fd, 0));

                int result_fd = open(sol_path.c_str(), O_CREAT | O_RDWR, 0644);
		ftruncate(result_fd, sizeof(float) * size);
		float *result = reinterpret_cast<float *>(mmap(NULL, sizeof(float) * size, PROT_WRITE | PROT_READ, MAP_SHARED, result_fd, 0));

                float *img_d, *kernel_d, *result_d;
                hipMalloc((void**)&img_d, size);
                hipMemcpy(img_d, img, size, hipMemcpyHostToDevice);

                hipMalloc((void**)&kernel_d, 9 * sizeof(float));
                hipMemcpy(kernel_d, kernel, 9 * sizeof(float), hipMemcpyHostToDevice);

                hipMalloc((void **) &result_d, size);

                convolution2D<<<1, 1>>>(img_d, kernel_d, result_d, size);

                hipDeviceSynchronize();

                hipMemcpy(result, result_d, size, hipMemcpyDeviceToHost);
                return sol_path;
        }
};
